#include <iostream>
#include <fstream>
#include <unistd.h>

#include <sys/time.h>
#include "tester.h"
#include "include/time_utils.h"
#include "include/utils.h"
#include "../distributed_mmio/include/mmio.h"

#define eprintf(...) fprintf (stderr, __VA_ARGS__)

using std::cout, std::endl;

int timed_main(const char*);


int main(const int argc, char** argv)
{
    int ret = 1;
    TIMER_DEF(1);
    srand(time(nullptr));

    TIMER_START(1);
    if (argc >= 2)
        ret = timed_main(argv[1]);
    else
        cout << "FATAL: require filename argument" << endl;
    TIMER_STOP(1);

    cout << "TOTAL PROGRAM TIME: " << TIMER_ELAPSED_MS(1) << "ms" << endl;
    return ret;
}


int timed_main(const char* input_file)
{
    cout << "\n* Started" << endl;
    TIMER_DEF(0);
    TIMER_DEF(1);
    TIMER_DEF(2);
    TIMER_DEF(3);
    TIMER_DEF(4);
    TIMER_DEF(5);

    // Data allocation
    GpuCoo<uint32_t, float> matrix = {0, 0, 0, nullptr, nullptr, nullptr};
    float* vec = nullptr;
    float* res = nullptr;
    float* res_control = nullptr;

    // Reading matrix data
    TIMER_START(0);
    const COO_local<u32, float>* coo = Distr_MMIO_COO_local_read<u32, float>(input_file);
    if (coo == nullptr)
    {
        printf("Failed to import graph from file [%s]\n", input_file);
        return -1;
    }
    TIMER_STOP(0);
    cout << "* Read data" << endl;

    // Alloc memory
    TIMER_START(1);
    matrix.NON_ZERO = coo->nnz;
    matrix.COLS = coo->ncols;
    matrix.ROWS = coo->nrows;
    hipMallocManaged(&matrix.xs, matrix.NON_ZERO * sizeof(int));
    hipMallocManaged(&matrix.ys, matrix.NON_ZERO * sizeof(int));
    hipMallocManaged(&matrix.vals, matrix.NON_ZERO * sizeof(float));

    // Alloc memory for other part
    hipMallocManaged(&vec, matrix.COLS * sizeof(float));
    hipMallocManaged(&res, matrix.ROWS * sizeof(float));
    hipMallocManaged(&res_control, matrix.ROWS * sizeof(float));
    TIMER_STOP(1);
    cout << "* Allocated  memory" << endl;

    // Copy data to GPU
    TIMER_START(2);
    hipMemcpy(matrix.xs, coo->col, matrix.NON_ZERO * sizeof(u32), hipMemcpyHostToDevice);
    hipMemcpy(matrix.ys, coo->row, matrix.NON_ZERO * sizeof(u32), hipMemcpyHostToDevice);
    hipMemcpy(matrix.vals, coo->val, matrix.NON_ZERO * sizeof(float), hipMemcpyHostToDevice);
    TIMER_STOP(2);
    cout << "* Copied COO to GPU memory" << endl;

    // Generation of random vector
    TIMER_START(3);
    randomize_dense_vec(vec, matrix.ROWS);
    TIMER_STOP(3);
    cout << "* Randomized Vector" << endl;

    // Execution
    TIMER_START(4);
    execution(matrix, vec, res, res_control);
    TIMER_STOP(4);
    cout << "* Terminated execution" << endl;

    // Free memory and close resources
    TIMER_START(5);
    hipFree(matrix.xs);
    hipFree(matrix.ys);
    hipFree(matrix.vals);
    hipFree(vec);
    hipFree(res_control);
    hipFree(res);
    TIMER_STOP(5);
    cout << "* Finished Deallocating\n" << endl;

    // Print time
    cout << "Time elapsed for reading: " << TIMER_ELAPSED_MS(0) << " ms" << endl;
    cout << "Time elapsed for allocation: " << TIMER_ELAPSED_MS(1) << " ms" << endl;
    cout << "Time elapsed for copy: " << TIMER_ELAPSED_MS(2) << " ms" << endl;
    cout << "Time elapsed for vector generation: " << TIMER_ELAPSED_MS(3) << " ms" << endl;
    cout << "Time elapsed for full tester: " << TIMER_ELAPSED_MS(4) << " ms" << endl;
    cout << "Time elapsed for free : " << TIMER_ELAPSED_MS(5) << " ms" << endl;
    return 0;
}
