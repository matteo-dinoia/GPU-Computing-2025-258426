#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include "include/type_alias.h"
#include "include/utils.h"

using std::cout, std::endl;

#define RAND_BOUND 10
#define RAND_PREC 1000

void randomize_dense_vec(MV* vec, const u32 N)
{
    for (u32 i = 0; i < N; i++)
    {
        const auto tmp = static_cast<MV>(rand() % (RAND_BOUND * RAND_PREC * 2)); // NOLINT(*-msc50-cpp)
        vec[i] = tmp / RAND_PREC - RAND_BOUND;
    }
}

void print_diff_info(const MV* v, const MV* control, const u32 LEN, const std::string_view name)
{
    u32 n_error = 0;
    u32 i_first_err = 0;

    for (u32 i = 0; i < LEN; i++)
    {
        if (std::fabs(v[i] - control[i]) > std::max(std::fabs(control[i] * 0.1), 0.01))
        {
            if (n_error == 0)
                i_first_err = i;
            n_error++;
        }
    }

    if (n_error > 0)
    {
        cout << "ERROR/s in " << name << " there are " << n_error << " over " << LEN << " [ first at index "
             << i_first_err << " where found " << v[i_first_err] << " insted of expected " << control[i_first_err]
             << "]" << endl;

        if (LEN < 20)
        {
            PRINT_VEC(v, LEN);
            PRINT_VEC(control, LEN);
        }
    }
}

void print_min_max(const MV* v, const u32 len)
{
    MV max = -100;
    MV min = 100;

    for (u32 i = 0; i < len; i++)
    {
        max = std::max(v[i], max);
        min = std::min(v[i], min);
    }

    cout << "Vector is in range [" << min << ", " << max << "]" << endl;
}

bool is_sorted_indexes(const u32* v, const u32 len)
{
    for (u32 i = 1; i < len; i++)
    {
        if (v[i - 1] > v[i])
            return false;
    }
    return true;
}

u32 lowest_greater_2_power(const u32 n)
{
    u32 res = 1;
    while (res < n)
        res <<= 1;
    return res;
}
