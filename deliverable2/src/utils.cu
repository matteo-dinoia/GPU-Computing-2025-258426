#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include "include/type_alias.h"
#include "include/utils.h"

using std::cout, std::endl;

#define RAND_BOUND 10
#define RAND_PREC 1000

void randomize_dense_vec(MV* vec, const MI N)
{
    for (MI i = 0; i < N; i++)
    {
        const auto tmp = static_cast<MV>(rand() % (RAND_BOUND * RAND_PREC * 2)); // NOLINT(*-msc50-cpp)
        vec[i] = tmp / RAND_PREC - RAND_BOUND;
    }
}

void print_diff_info(const MV* v, const MV* control, const MI LEN, const std::string_view name)
{
    MI n_error = 0;
    MI i_first_err = 0;
    MI n_warning = 0;

    for (MI i = 0; i < LEN; i++)
    {
        if (std::fabs(v[i] - control[i]) > std::max(std::fabs(control[i] * 2.0), 0.5))
        {
            if (n_error == 0)
                i_first_err = i;
            n_error++;
        }
        else if (std::fabs(v[i] - control[i]) > std::max(std::fabs(control[i] * 0.25), 0.1))
        {
            if (n_error == 0 && n_warning == 0)
                i_first_err = i;
            n_warning++;
        }

    }

    if (n_error > 0)
    {
        cout << "[!] ERROR/s in " << name << " there are " << n_error << " errors and " << n_warning <<
            " warnings over " << LEN << " [ first at index "
            << i_first_err << " where found " << v[i_first_err] << " insted of expected " << control[i_first_err]
            << "]" << endl;
    }
    else if (n_warning > 0)
    {
        cout << "[?] WARNING/s in " << name << " there are " << n_warning << " warnings over " << LEN <<
            " [ first at index "
            << i_first_err << " where found " << v[i_first_err] << " insted of expected " << control[i_first_err]
            << "]" << endl;
    }

    if (LEN < 20 && (n_error > 0 || n_warning > 0))
    {
        PRINT_VEC(v, LEN);
        PRINT_VEC(control, LEN);
    }
}

void print_min_max(const MV* v, const MI len)
{
    MV max = -100;
    MV min = 100;

    for (MI i = 0; i < len; i++)
    {
        max = std::max(v[i], max);
        min = std::min(v[i], min);
    }

    cout << "Vector is in range [" << min << ", " << max << "]" << endl;
}

bool is_sorted_indexes(const MI* v, const MI len)
{
    for (MI i = 1; i < len; i++)
    {
        if (v[i - 1] > v[i])
            return false;
    }
    return true;
}

MI lowest_greater_2_power(const MI n)
{
    MI res = 1;
    while (res < n)
        res <<= 1;
    return res;
}
